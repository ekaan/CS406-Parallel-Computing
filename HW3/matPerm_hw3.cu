
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>

using namespace std;



#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
  if(e!=hipSuccess) {                                              \
     printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
        exit(0); \
	 }                                                                 \
	 }
  
void usage()
{
  cout << "USAGE: ./exec <filename>" << endl;
  exit(0);
}

__global__
void perMat (double* x, double* x_pre_d, int rowSize, long long* myMat, double* p, long long tNum) {
  unsigned long long tn11 = 1LL << (rowSize-1);
  unsigned long long chunkSize = tn11 / tNum;
  
  unsigned long long t_Id = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long start_loc = t_Id * chunkSize + 1;
  unsigned long long end_loc = min(start_loc + chunkSize, tn11);

  double *my_x =  x + t_Id*rowSize;
  for (int i = 0 ; i < rowSize ; i++) {
    my_x[i] = x_pre_d[i];
  }

  long long gray = (start_loc - 1) ^ ((start_loc-1) >> 1);
  int ct = 0;
  while(gray) {
    if(gray & 1LL) {
      for(int k = 0 ; k < rowSize ; k++) {
        my_x[k] += myMat[ct*rowSize + k];
      }
    }
    gray = gray >> 1;
    ct += 1;
  }
  
  
  double local_p = 0;

  if(t_Id < 16) {
    printf("%d, %d, %d, %ld, %ld\n",  blockIdx.x, blockDim.x, threadIdx.x, start_loc, end_loc);
  }
  
  int pr_Sign = 1; if(start_loc % 2 == 1) pr_Sign = -1;  
  for (long long i = start_loc; i < end_loc; i++) {
    unsigned long long prev_gray, bit_id;
    
    gray = (i >> 1) ^ i;
    prev_gray = ((i-1) >> 1) ^ (i-1);
    bit_id = __ffs(gray ^ prev_gray) - 1;// -1

    int s = -1;
    if(gray & (1LL << bit_id)) {
      s = 1;
    }
        
    double prod_x = 1;
    for(int j = 0 ; j < rowSize ; j++) {
      my_x[j] += myMat[bit_id * rowSize + j] * s;
      prod_x *= my_x[j];
    }
    local_p += pr_Sign * prod_x;
    pr_Sign *= -1;
  }
  atomicAdd(p, local_p);
}

int main(int argc, const char** argv)
{

  if(argc != 3)
    usage();

  string line;

  const char* filename = argv[1];
  int devId = atoi(argv[2]);

  ifstream input (filename);
  if(input.fail())
    return 0;


  long long N;
  int **M;
  getline(input,line);
  N = atoi(line.c_str());

  M = new int*[N];
  for(int i = 0; i < N; i ++)
    M[i] = new int[N];


  int linectr = 0;
  while(getline(input,line)){
    stringstream ss(line);
    int temp;
    int ctr = 0;
    while(ss >> temp)
      M[linectr][ctr++] = temp;

    linectr++;
  }

  double start, end;
  ////YOUR CODE GOES HERE/////----------------------------------------------

  hipSetDevice(devId); // SET DEVICE

  long long tn11 = (long long)(( (long long)1 << (N-1) ));

  cout << "tn11: " << tn11 << endl;

  int ** M_T = new int* [N];
  for(int j = 0 ; j < N ; j++)
  {
    M_T[j] = new int [N];
    for(int k = 0 ; k < N ; k++)
    {
      M_T[j][k] = M[k][j];
    }
  }

  long long * oneDMat = new long long [N*N];

  for (int i = 0 ; i < N ; i++)
    for (int k = 0 ; k < N ; k++)
      oneDMat[i*N+k] = M_T[i][k];

  ////CONVERTING MATRIX TO ARRAY/////----------------------------------------

  double p = 1;
  double lastColumn, t_sum;

  double* x_pre_h = new double [N];
  for(int i = 0 ; i < N ; i++)
  {
    //lastColumn = M[i][N-1]; //--without transpose
    lastColumn = M_T[N-1][i];
    t_sum = 0;

    for(int j = 0 ; j < N ; j++)
      t_sum += M[i][j];  // M normally

    x_pre_h[i] = lastColumn - t_sum/2.0;
    p *= x_pre_h[i];
  }

  long long th_num;
  dim3 my_block,my_grid;
  //FINE TUNING
  if(tn11 == ((long long)1 << 14 ))
  {
    dim3 block(128,1,1);
    my_block = block;
    dim3 grid(128,1,1);
    my_grid = grid;
    th_num = ( (long long)1 << 14 );
  }
  else if(tn11 == ((long long)1 << 19 ))
  {
    dim3 block(512,1,1);
    my_block = block;
    dim3 grid(1024,1,1);
    my_grid = grid;
    th_num = ( (long long)1 << 19 );
  }
  else
  {
    dim3 block(1024,1,1);
    my_block = block;
    dim3 grid(1024,1,1);
    my_grid = grid;
    th_num = ( (long long)1 << 20 );
  }

  cout << "Thread num: " << th_num << endl;
  cout << "tn11: " << tn11 << endl;
  cout << "Chunk size: " << tn11 / th_num << endl;
/*
  long long* chunk_ptr;
  cudaMalloc((void**)&chunk_ptr, sizeof(long long));
  cudaMemcpy(chunk_ptr, &chunk, sizeof(long long), cudaMemcpyHostToDevice);
*/
/*
  long long* tn11_ptr;
  cudaMalloc((void**)&tn11_ptr, sizeof(long long));
  cudaMemcpy(tn11_ptr, &tn11, sizeof(long long), cudaMemcpyHostToDevice);
*/
  double* x_pre_d;
  hipMalloc((void**)&x_pre_d, N*sizeof(double));
  hipMemcpy(x_pre_d, x_pre_h, sizeof(double)*N, hipMemcpyHostToDevice);

  double* x = new double [N*th_num];
  double* x_k;
  hipMalloc((void**)&x_k, th_num*N*sizeof(double));
  //  cudaMemcpy(x_k, x, sizeof(double)*N*th_num, cudaMemcpyHostToDevice);

  double* p_ptr;
  hipMalloc((void**)&p_ptr, sizeof(double));
  hipMemcpy(p_ptr, &p, sizeof(double), hipMemcpyHostToDevice);

  long long* myMat;
  hipMalloc((void**)&myMat, sizeof(long long)*N*N);
  hipMemcpy(myMat, oneDMat, sizeof(long long)*N*N, hipMemcpyHostToDevice);

  //// ITERATION STARTS HERE/////-------------------------------------------
  start = omp_get_wtime();


  for (int i = 0 ; i < N ; i++) {
    for (int k = 0 ; k < N ; k++) {
      cout << oneDMat[i * N + k] << " ";
    }
    cout << endl;
  }
  
  //perMat<<32,1024>>(x, x_pre_h, N, oneDMat, p, tn11, chunk);
  perMat<<<my_grid,my_block>>>(x_k, x_pre_d, N, myMat, p_ptr, th_num);
  hipDeviceSynchronize();
  cudaCheckError();
  cout << "1 | " << p << endl;

  hipMemcpy(&p, p_ptr, sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  cout << "2 | " << p << endl;

  double result = (4 * (N & 1) - 2) * p;
  //// ITERATION ENDS HERE/////---------------------------------------------
 end = omp_get_wtime();
 cout << "Threads: " << th_num << "\tResult:" << result << "\tTime:" << end - start << " s" << endl;

  hipFree(x_k);
  hipFree(myMat);
  hipFree(x_pre_d);
  hipFree(p_ptr);

   return 0;
  }
